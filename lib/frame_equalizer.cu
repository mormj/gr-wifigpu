#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>


__host__ __device__ double carg(const hipFloatComplex &z) {
  return atan2(hipCimagf(z), hipCrealf(z));
}
__host__ __device__ hipFloatComplex conj(const hipFloatComplex &z) {
  return make_hipFloatComplex(z.x, -z.y);
}

__global__ void calc_beta_err_kernel(hipFloatComplex *in, float *polarity,
                                     int current_symbol_index,
                                     hipFloatComplex *last_symbol, float bw,
                                     float freq, float *beta, float *err,
                                     int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n) {
    hipFloatComplex *current_symbol = &in[i*64];
    hipFloatComplex pp0, pp1, pp2, pp3;

   
    if (i > 0) {
      pp0 = in[(i - 1)*64 + 11];
      pp1 = in[(i - 1)*64 + 25];
      pp2 = in[(i - 1)*64 + 39];
      pp3 = in[(i - 1)*64 + 53];
    }
    else {
      pp0 = last_symbol[11];
      pp1 = last_symbol[25];
      pp2 = last_symbol[39];
      pp3 = last_symbol[53];
    }

    float p = polarity[(current_symbol_index + i - 2) % 127];

		if(current_symbol_index+i <= 2) {
      pp1 = hipCmulf(pp1, make_hipFloatComplex(-1.0,0.0));
		} else {
      float last_p = polarity[(current_symbol_index + i - 2 - 1) % 127];
			pp0 = hipCmulf(pp0, make_hipFloatComplex(last_p,0.0));
			pp1 = hipCmulf(pp1, make_hipFloatComplex(last_p,0.0));
			pp2 = hipCmulf(pp2, make_hipFloatComplex(last_p,0.0));
			pp3 = hipCmulf(pp3, make_hipFloatComplex(-last_p,0.0));
		}

    if ((current_symbol_index + i) < 2) {
      beta[i] = carg(
          make_hipFloatComplex(current_symbol[11].x - current_symbol[25].x +
                                  current_symbol[39].x + current_symbol[53].x,
                              current_symbol[11].y - current_symbol[25].y +
                                  current_symbol[39].y + current_symbol[53].y));
    } else {
      beta[i] = carg(make_hipFloatComplex(
          (current_symbol[11].x * p) + (current_symbol[39].x * p) +
              (current_symbol[25].x * p) + (current_symbol[53].x * -p),
          (current_symbol[11].y * p) + (current_symbol[39].y * p) +
              (current_symbol[25].y * p) + (current_symbol[53].y * -p)));

    }

    err[i] = carg(hipCaddf(
        hipCaddf((make_hipFloatComplex(
                    hipCmulf(pp0, current_symbol[11]).x * p,
                    -hipCmulf(pp0, current_symbol[11]).y * p)),
                (make_hipFloatComplex(
                    hipCmulf(pp1, current_symbol[25]).x * p,
                    -hipCmulf(pp1, current_symbol[25]).y * p))),
        hipCaddf((make_hipFloatComplex(
                    hipCmulf(pp2, current_symbol[39]).x * p,
                    -hipCmulf(pp2, current_symbol[39]).y * p)),
                (make_hipFloatComplex(
                    hipCmulf(pp3, current_symbol[53]).x * -p,
                    -hipCmulf(pp3, current_symbol[53]).y * -p)))));

    err[i] *= (bw / (2 * M_PI * freq * 80));
  }
}

void exec_calc_beta_err(hipFloatComplex *in, float *polarity,
                        int current_symbol_index, hipFloatComplex *last_symbol,
                        float bw, float freq, float *beta, float *err, int n,
                        int grid_size, int block_size, hipStream_t stream) {
  calc_beta_err_kernel<<<grid_size, block_size, 0, stream>>>(
      in, polarity, current_symbol_index, last_symbol, bw, freq, beta, err, n);
}

void get_block_and_grid_calc_beta_err(int *minGrid, int *minBlock) {
  hipOccupancyMaxPotentialBlockSize(minGrid, minBlock, calc_beta_err_kernel, 0,
                                     0);
}