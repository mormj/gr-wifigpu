#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>


__global__ void multiply_const_kernel(hipFloatComplex *in, hipFloatComplex *out,
                                      hipFloatComplex k, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    // e ix = cos x + i sin x
    out[i] = hipCmulf(in[i], k);
  }
}

__global__ void multiply_phase_kernel(hipFloatComplex *in, hipFloatComplex *out,
                                      float beta, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    // e ix = cos x + i sin x
    out[i] = hipCmulf( in[i], make_hipFloatComplex(cos(beta), sin(beta)));
  }
}

void exec_multiply_const(hipFloatComplex *in, hipFloatComplex *out,
                         hipFloatComplex k, int n, int grid_size, int block_size,
                         hipStream_t stream) {
  multiply_const_kernel<<<grid_size, block_size, 0, stream>>>(in, out, k, n);
}

void exec_multiply_phase(hipFloatComplex *in, hipFloatComplex *out, float beta,
                         int n, int grid_size, int block_size,
                         hipStream_t stream) {
  multiply_phase_kernel<<<grid_size, block_size, 0, stream>>>(in, out, beta, n);
}

void get_block_and_grid_multiply_const(int *minGrid, int *minBlock) {
  hipOccupancyMaxPotentialBlockSize(minGrid, minBlock, multiply_const_kernel,
                                     0, 0);
}